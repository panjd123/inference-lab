#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>

__global__ void square_array_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx] * input[idx];
    }
}

extern "C" void native_cuda_square_array(const float* input, float* output, int size) {
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    square_array_kernel<<<numBlocks, blockSize>>>(input, output, size);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
}

extern "C" void cuda_square_array(const float* input, float* output, int size) {
    float* d_input;
    float* d_output;
    size_t bytes = size * sizeof(float);
    hipMalloc((void**)&d_input, bytes);
    hipMalloc((void**)&d_output, bytes);
    hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);
    native_cuda_square_array(d_input, d_output, size);
    hipMemcpy(output, d_output, bytes, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
}

double cuda_benchmark(void (*func)(const float*, float*, int), const float* input, float* output, int size, int iter) {
    float* d_input;
    float* d_output;
    size_t bytes = size * sizeof(float);
    hipMalloc((void**)&d_input, bytes);
    hipMalloc((void**)&d_output, bytes);
    hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < iter; i++) {
        func(d_input, d_output, size);
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    double ms = std::chrono::duration<double, std::milli>(end - start).count();
    hipMemcpy(output, d_output, bytes, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
    hipDeviceReset();
    return ms / iter;
}